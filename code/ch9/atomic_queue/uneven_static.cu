// uneven_static.cu
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void computeKernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int work = idx % 256;
        float result = 0.0f;
        for (int i = 0; i < work; ++i) {
            result += sinf(input[idx]) * cosf(input[idx]);
        }
        output[idx] = result;
    }
}

int main() {
    const int N = 1<<20;
    float *h_in = new float[N], *h_out = new float[N];
    for (int i = 0; i < N; ++i) h_in[i] = float(i)/N;
    float *d_in, *d_out;
    hipMalloc(&d_in, N*sizeof(float));
    hipMalloc(&d_out, N*sizeof(float));
    hipMemcpy(d_in, h_in, N*sizeof(float), hipMemcpyHostToDevice);
    dim3 block(256), grid((N+255)/256);
    computeKernel<<<grid, block>>>(d_in, d_out, N);
    hipDeviceSynchronize();
    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
