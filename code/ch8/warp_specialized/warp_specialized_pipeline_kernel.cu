#include "hip/hip_runtime.h"
#include <cuda/pipeline>
#include <cooperative_groups>
#include <hip/hip_runtime.h>
namespace cg = cooperative_groups;
#define TILE_SIZE 1024

__device__ float computeTile(const float* data, int lane) {
    return data[lane]*1.0f;
}

__global__ void warp_specialized_pipeline_kernel(const float* __restrict__ A,
                                                 const float* __restrict__ B,
                                                 float* __restrict__ C,
                                                 int nTiles) {
    cg::thread_block block = cg::this_thread_block();
    extern __shared__ float shared_mem[];
    float* buf = shared_mem;
    cuda::pipeline<3> pipe(block);
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int warp_id = tid>>5, lane = tid&31;
    int totalWarps = (gridDim.x*blockDim.x)>>5;
    for(int t=warp_id; t<nTiles; t+=totalWarps) {
        size_t offset=t*TILE_SIZE;
        if(warp_id==0){
            pipe.producer_acquire(0);
            __pipeline_memcpy_async(buf+lane, A+offset+lane, TILE_SIZE*sizeof(float));
            pipe.producer_commit(0);
        }
        if(warp_id==1){
            pipe.consumer_wait(0);
            pipe.producer_acquire(1);
            float v=computeTile(buf, lane);
            buf[lane]=v;
            pipe.producer_commit(1);
            pipe.consumer_release(0);
        }
        if(warp_id==2){
            pipe.consumer_wait(1);
            pipe.producer_acquire(2);
            C[offset+lane]=buf[lane];
            pipe.producer_commit(2);
            pipe.consumer_release(1);
        }
    }
}

int main(){
    const int nTiles=4;
    float *dA,*dB,*dC;
    hipMalloc(&dA,nTiles*TILE_SIZE*sizeof(float));
    hipMalloc(&dB,nTiles*TILE_SIZE*sizeof(float));
    hipMalloc(&dC,nTiles*TILE_SIZE*sizeof(float));
    warp_specialized_pipeline_kernel<<<1,96, TILE_SIZE*sizeof(float)>>>(dA,dB,dC,nTiles);
    hipDeviceSynchronize();
    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}